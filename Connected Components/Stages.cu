#include "hip/hip_runtime.h"
#include ""
#include "Stages.cuh"
#include "CudaUtils.cuh"

namespace sv {

	__device__ bool has_grafted_d = true;

	__global__ void InitComponentKernel(int *component, const int numVertices)
	{
		const auto tid = blockIdx.x*blockDim.x + threadIdx.x;
		const auto numThreads = gridDim.x * blockDim.x;

		for (auto i = tid; i < numVertices; i += numThreads) {
			component[i] = i;
		}
	}

	__global__ void GraftKernel(std::pair<int, int> *graph, const int numEdges, int *component)
	{

		const auto tid = blockIdx.x*blockDim.x + threadIdx.x;
		const auto numThreads = gridDim.x * blockDim.x;

		has_grafted_d = false;

		for (auto i = tid; i < numEdges; i += numThreads)
		{

			int fromVertex = graph[i].first;
			int toVertex = graph[i].second;

			int fromComponent = component[fromVertex];
			int toComponent = component[toVertex];

			if ((fromComponent < toComponent) && (toComponent == component[toComponent]))
			{
				has_grafted_d = true;
				component[toComponent] = fromComponent;

			}


			const int tmp = fromVertex;
			fromVertex = toVertex;
			toVertex = tmp;

			fromComponent = component[fromVertex];
			toComponent = component[toVertex];

			if ((fromComponent < toComponent) && (toComponent == component[toComponent]))
			{
				has_grafted_d = true;
				component[toComponent] = fromComponent;
			}
		}
	}

	__global__ void ShortcutKernel(int *component, const int numVertices)
	{

		const auto tid = blockIdx.x*blockDim.x + threadIdx.x;
		const auto numThreads = gridDim.x * blockDim.x;

		for (auto i = tid; i < numVertices; i += numThreads)
		{
			while (component[i] != component[component[i]])
			{
				component[i] = component[component[i]];
			}
		}
	}

}


namespace svu {

	__device__ bool has_grafted_d = false;

	__global__ void InitComponentKernel(int *component, const int numVertices)
	{
		const auto tid = blockIdx.x*blockDim.x + threadIdx.x;
		const auto numThreads = gridDim.x * blockDim.x;

		for (auto i = tid; i < numVertices; i += numThreads) {
			component[i] = i;
		}
	}

	__global__ void GraftKernel(std::pair<int, int> *graph, const int numEdges, int *component)
	{

		const auto tid = blockIdx.x*blockDim.x + threadIdx.x;
		const auto numThreads = gridDim.x * blockDim.x;

		has_grafted_d = false;

		for (auto i = tid; i < numEdges; i += numThreads)
		{

			int fromVertex = graph[i].first;
			int toVertex = graph[i].second;

			if (fromVertex < toVertex)
			{
				has_grafted_d = true;
				component[toVertex] = fromVertex;
			}

			const int tmp = fromVertex;
			fromVertex = toVertex;
			toVertex = tmp;

			if (fromVertex < toVertex)
			{
				has_grafted_d = true;
				component[toVertex] = fromVertex;
			}
		}
	}

	__global__ void ShortcutKernel(int *component, const int numVertices)
	{

		const auto tid = blockIdx.x*blockDim.x + threadIdx.x;
		const auto numThreads = gridDim.x * blockDim.x;

		for (auto i = tid; i < numVertices; i += numThreads)
		{
			while (component[i] != component[component[i]])
			{
				component[i] = component[component[i]];
			}
		}
	}

	__global__ void UpdateKernel(std::pair<int, int> *graph, const int numEdges, int *component)
	{
		const auto tid = blockIdx.x*blockDim.x + threadIdx.x;
		const auto numThreads = gridDim.x * blockDim.x;

		for (auto i = tid; i < numEdges; i += numThreads)
		{
			graph[i].first = component[graph[i].first];
			graph[i].second = component[graph[i].second];
		}
	}
}


void ShiloachVishkinUpdt(std::pair<int, int> *graph, const int numEdges, int *components, const int numVertices)
{

	//Set caching prefrence, prefer L1
	CHECK(hipDeviceSetCacheConfig(hipFuncCachePreferL1));

	// init device memory
	std::pair<int, int> *d_graph = nullptr;
	int *d_results = nullptr;

	const int numBytesGraph = numEdges * sizeof(std::pair<int, int>);
	const int numBytesResult = numVertices * sizeof(int);


	CHECK(hipMalloc((void **)&d_graph, numBytesGraph));
	CHECK(hipMalloc((void **)&d_results, numBytesResult));

	// has grafted
	bool has_grafted_h = true;


	// transfer data from host to device
	CHECK(hipMemcpy(d_graph, graph, numBytesGraph, hipMemcpyHostToDevice));


	int threads_per_block = 1024;
	int blocks_per_grid = 30;

	svu::InitComponentKernel << <blocks_per_grid, threads_per_block >> > (d_results, numVertices);
	//synchronize with host
	CHECK(hipDeviceSynchronize());
	// check for errors
	CHECK(hipGetLastError());

	while (has_grafted_h)
	{
		// execute graft kernel
		svu::GraftKernel << <blocks_per_grid, threads_per_block >> > (d_graph, numEdges, d_results);
		//synchronize with host
		CHECK(hipDeviceSynchronize());
		// check for errors
		CHECK(hipGetLastError());
		// execute shortcut kernel
		svu::ShortcutKernel << <blocks_per_grid, threads_per_block >> > (d_results, numVertices);
		//synchronize with host
		CHECK(hipDeviceSynchronize());
		// check for errors
		CHECK(hipGetLastError());
		// execute update kernel
		svu::UpdateKernel << <blocks_per_grid, threads_per_block >> > (d_graph, numEdges, d_results);
		//synchronize with host
		CHECK(hipDeviceSynchronize());
		// check for errors
		CHECK(hipGetLastError());
		// check if has grafted
		CHECK(hipMemcpyFromSymbol(&has_grafted_h, HIP_SYMBOL(svu::has_grafted_d), sizeof(bool), 0, hipMemcpyDeviceToHost));
	}

	// copy kernel result back to host side
	CHECK(hipMemcpy(components, d_results, numBytesResult, hipMemcpyDeviceToHost));


	// free device memory
	CHECK(hipFree(d_graph));
	CHECK(hipFree(d_results));
}


void ShiloachVishkin(std::pair<int, int> *graph, const int numEdges, int *components, const int numVertices)
{

	//Set caching prefrence, prefer L1
	CHECK(hipDeviceSetCacheConfig(hipFuncCachePreferL1));

	// init device memory
	std::pair<int, int> *d_graph = nullptr;
	int *d_results = nullptr;

	const int numBytesGraph = numEdges * sizeof(std::pair<int, int>);
	const int numBytesResult = numVertices * sizeof(int);

	CHECK(hipMalloc((void **)&d_graph, numBytesGraph));
	CHECK(hipMalloc((void **)&d_results, numBytesResult));

	// has grafted
	bool has_grafted_h = true;

	// transfer data from host to device
	CHECK(hipMemcpy(d_graph, graph, numBytesGraph, hipMemcpyHostToDevice));

	// max number of blocks per dim = 65535
	// max number of threads per block = 1024
	// figure out optimal block size, and gridsize
	// (numEdges + threadsPerBlock.x - 1) / threadsPerBlock.x

	// (15) Multiprocessors, (128) CUDA Cores/MP:     1920 CUDA Cores
	// Maximum number of threads per multiprocessor:  2048
	// Maximum number of threads per block:           1024
	// Max dimension size of a thread block(x, y, z): (1024, 1024, 64)
	// Max dimension size of a grid size(x, y, z):    (2147483647, 65535, 65535)

	int threads_per_block = 1024;
	int blocks_per_grid = 30;

	sv::InitComponentKernel << <blocks_per_grid, threads_per_block >> > (d_results, numVertices);
	//synchronize with host
	CHECK(hipDeviceSynchronize());
	// check for errors
	CHECK(hipGetLastError());

	while (has_grafted_h)
	{
		// execute graft kernel
		sv::GraftKernel << <blocks_per_grid, threads_per_block >> > (d_graph, numEdges, d_results);
		//synchronize with host
		CHECK(hipDeviceSynchronize());
		// check for errors
		CHECK(hipGetLastError());
		// execute shortcut kernel
		sv::ShortcutKernel << <blocks_per_grid, threads_per_block >> > (d_results, numVertices);
		//synchronize with host
		CHECK(hipDeviceSynchronize());
		// check for errors
		CHECK(hipGetLastError());
		// check if hasGrafted
		CHECK(hipMemcpyFromSymbol(&has_grafted_h, HIP_SYMBOL(sv::has_grafted_d), sizeof(bool), 0, hipMemcpyDeviceToHost));
	}

	// copy kernel result back to host side
	CHECK(hipMemcpy(components, d_results, numBytesResult, hipMemcpyDeviceToHost));

	// free device memory
	CHECK(hipFree(d_graph));
	CHECK(hipFree(d_results));
}

void Stages(std::pair<int, int>* graph, const int numEdges, int * components, const int numVertices)
{


	//randomly permute El
	//std::random_device randomDevice;
	//std::mt19937 engine(randomDevice());
	//std::shuffle(graph, graph + numEdges, engine);


	// divide El into groups
	int numberOfGroups = 1;
	int numberofEdgesPerGroup = numEdges;
	const double q = 0.6;
	const double numberOfEdgesPerVertex = numEdges / static_cast<double>(numVertices);

	
	if (q < numberOfEdgesPerVertex)
	{
		numberofEdgesPerGroup = q*numVertices;
		numberOfGroups = ceil(numEdges / static_cast<double>(numberofEdgesPerGroup));
	}

	//Set caching prefrence, prefer L1
	CHECK(hipDeviceSetCacheConfig(hipFuncCachePreferL1));

	// init device memory
	std::pair<int, int> *d_graph = nullptr;
	int *d_results = nullptr;

	const int numBytesGraph = numEdges * sizeof(std::pair<int, int>);
	const int numBytesResult = numVertices * sizeof(int);


	CHECK(hipMalloc((void **)&d_graph, numBytesGraph));
	CHECK(hipMalloc((void **)&d_results, numBytesResult));

	// transfer data from host to device
	CHECK(hipMemcpy(d_graph, graph, numBytesGraph, hipMemcpyHostToDevice));


	int threads_per_block = 1024;
	int blocks_per_grid = 30;

	// initialize components
	svu::InitComponentKernel << <blocks_per_grid, threads_per_block >> > (d_results, numVertices);
	// synchronize with host
	CHECK(hipDeviceSynchronize());
	// check for errors
	CHECK(hipGetLastError());

	

	for (int i = 0; i < numberOfGroups; ++i)
	{
		const int groupStart = numberofEdgesPerGroup*i;
		int currentNumberofEdgesPerGroup = numberofEdgesPerGroup;
		if (i == numberOfGroups - 1) { currentNumberofEdgesPerGroup = numEdges - (numberOfGroups - 1)*numberofEdgesPerGroup; }

		// has grafted
		bool has_grafted_h = true;

		while (has_grafted_h)
		{
			// execute graft kernel
			svu::GraftKernel << <blocks_per_grid, threads_per_block >> > (d_graph + groupStart, currentNumberofEdgesPerGroup, d_results);
			//synchronize with host
			CHECK(hipDeviceSynchronize());
			// check for errors
			CHECK(hipGetLastError());
			// execute shortcut kernel
			svu::ShortcutKernel << <blocks_per_grid, threads_per_block >> > (d_results, numVertices);
			//synchronize with host
			CHECK(hipDeviceSynchronize());
			// check for errors
			CHECK(hipGetLastError());
			// execute update kernel
			svu::UpdateKernel << <blocks_per_grid, threads_per_block >> > (d_graph + groupStart, currentNumberofEdgesPerGroup, d_results);
			//synchronize with host
			CHECK(hipDeviceSynchronize());
			// check for errors
			CHECK(hipGetLastError());
			// check if has grafted
			CHECK(hipMemcpyFromSymbol(&has_grafted_h, HIP_SYMBOL(svu::has_grafted_d), sizeof(bool), 0, hipMemcpyDeviceToHost));
		}


		if (i < (numberOfGroups - 1))
		{
			int numberOfEdgesNextGroup = currentNumberofEdgesPerGroup;
			// if second last group
			if(i==numberOfGroups-2) { numberOfEdgesNextGroup = numEdges - (numberOfGroups - 1)*numberofEdgesPerGroup; }
			// execute update kernel
			svu::UpdateKernel << <blocks_per_grid, threads_per_block >> > (d_graph + groupStart + currentNumberofEdgesPerGroup, numberOfEdgesNextGroup, d_results);
			//synchronize with host
			CHECK(hipDeviceSynchronize());
			// check for errors
			CHECK(hipGetLastError());
		}

	}

	// copy kernel result back to host side
	CHECK(hipMemcpy(components, d_results, numBytesResult, hipMemcpyDeviceToHost));

	// free device memory
	CHECK(hipFree(d_graph));
	CHECK(hipFree(d_results));

}